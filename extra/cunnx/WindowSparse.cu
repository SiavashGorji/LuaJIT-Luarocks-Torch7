#include "hip/hip_runtime.h"
#include "utils.h"
#define WINDOWSPARSE_THREADS 128
#define WINDOWSPARSE_STREAMS 8

__global__ void cunnx_WindowSparse_copyBiasOutput_kernel(
  float *output, const float** bias, int outputWindowSize)
{
  unsigned int k = blockIdx.x;
  const float *bias_k = bias[k];
  float *output_k = output + outputWindowSize*k;
  
  for (unsigned int i=threadIdx.x; i<outputWindowSize; i+=blockDim.x)
  {
    output_k[i] = bias_k[i];
  }
}

  
static int cunnx_WindowSparse_updateOutput(lua_State *L)
{ 
  /* input, inputIndice, outputIndice, gradOutput*/
  THCState *state = getCutorchState(L);
  // batchSize x inputWindowSize x inputSize
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  // batchSize
  THLongTensor *inputIndice = (THLongTensor*)luaT_checkudata(L, 3, "torch.LongTensor");
  THLongTensor *outputIndice = (THLongTensor*)luaT_checkudata(L, 4, "torch.LongTensor");
  
  int batchedGemmMax = luaT_getfieldcheckint(L, 1, "batchedGemmMax");
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int outputWindowSize = luaT_getfieldcheckint(L, 1, "outputWindowSize");
  int batchSize, inputWindowSize;
  
  // outputSize x inputSize
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  // outputSize
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  // batchSize
  THCharTensor *biasHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "inputHost", "torch.CharTensor");
  THCudaTensor *biasCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputCuda", "torch.CudaTensor");
  // batchSize x outputWindowSize
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_output", "torch.CudaTensor");
  
  THCudaTensor* output_, *weight_, *_weight_, *bias_, *input_;
  
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  
  float alpha = 1;
  float beta = 1;
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] <= inputSize, 2, "invalid input size"); 
  luaL_argcheck(L, inputIndice->nDimension == 1, 3, "1D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 1, 4, "1D(batch mode) tensor expected");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "Expecting contiguous input");
  
  batchSize = input->size[0];
  inputWindowSize = input->size[1];
  
  THCudaTensor_resize2d(state, output, input->size[0], outputWindowSize);
    
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    THError("CUBLAS initialization failed");
    
  output_ = THCudaTensor_new(state);
  weight_ = THCudaTensor_new(state);
  _weight_ = THCudaTensor_new(state);
  bias_ = THCudaTensor_new(state);
  input_ = THCudaTensor_new(state);
  
  /* copy bias into output */
  THCharTensor_resize1d(biasHost, batchSize*sizeof(float*));
  THCudaTensor_resize1d(state, biasCuda, batchSize*sizeof(float*)/sizeof(float));
  
  const float **biasB = (const float **)THCharTensor_data(biasHost);
  const float **biasB_d = (const float **)THCudaTensor_data(state, biasCuda);
  
  for (int i=0; i<batchSize; i++)
  {
    int outputIdx = THLongTensor_get1d(outputIndice, i) - 1;
    THCudaTensor_narrow(state, bias_, bias, 0, outputIdx, outputWindowSize);
    biasB[i] = THCudaTensor_data(state, bias_);
  }
  
  if(hipMemcpy(biasB_d, biasB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
    THError("hipMemcpy failed");
  
  /* call cudakernel */
  dim3 blocks(batchSize); // each cuda-block is an example
  dim3 threads(WINDOWSPARSE_THREADS);
  cunnx_WindowSparse_copyBiasOutput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(state, output), biasB_d, outputWindowSize
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
  
  if (sqrt(inputWindowSize*outputWindowSize) > batchedGemmMax)
  {
    hipStream_t streams[WINDOWSPARSE_STREAMS];
    
    for (int i=0; i<WINDOWSPARSE_STREAMS; i++)
    {
      if (hipStreamCreate(&streams[i]) != hipSuccess)
        THError("error initializing stream");
    }
    hipDeviceSynchronize();
    
    for (int i=0; i<batchSize; i++)
    {
      hipblasSetStream(handle, streams[i%WINDOWSPARSE_STREAMS]);
      
      int inputIdx = THLongTensor_get1d(inputIndice, i) - 1;
      int outputIdx = THLongTensor_get1d(outputIndice, i) - 1;
      
      THCudaTensor_select(state, output_, output, 0, i);
      THCudaTensor_select(state, input_, input, 0, i);
      THCudaTensor_narrow(state, _weight_, weight, 1, inputIdx, inputWindowSize);
      THCudaTensor_narrow(state, weight_, _weight_, 0, outputIdx, outputWindowSize);
      
      stat = hipblasSgemv(handle, HIPBLAS_OP_T,  inputWindowSize, outputWindowSize,
                        &alpha, (const float*)THCudaTensor_data(state, weight_), inputSize,
                        (const float*)THCudaTensor_data(state, input_), 1,
                        &beta, THCudaTensor_data(state, output_), 1);
    }
    
    hipblasSetStream(handle, NULL);
    hipDeviceSynchronize();
    
    for (int i=0; i<WINDOWSPARSE_STREAMS; i++)
    {
      if (hipStreamDestroy(streams[i]) != hipSuccess)
        THError("error destroying stream");
    }
    
    
  }
  else
  {  
    THCharTensor *inputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "inputHost", "torch.CharTensor");
    THCharTensor *weightHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "weightHost", "torch.CharTensor");
    THCharTensor *outputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "outputHost", "torch.CharTensor");
    
    THCudaTensor *inputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputCuda", "torch.CudaTensor");
    THCudaTensor *weightCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weightCuda", "torch.CudaTensor");
    THCudaTensor *outputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputCuda", "torch.CudaTensor");
    // put output back on top of the stack
    output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_output", "torch.CudaTensor");
    
    hipblasSetStream(handle, NULL);
    
    THCharTensor_resize1d(inputHost, batchSize*sizeof(float*));
    THCharTensor_resize1d(weightHost, batchSize*sizeof(float*));
    THCharTensor_resize1d(outputHost, batchSize*sizeof(float*));
    
    THCudaTensor_resize1d(state, inputCuda, batchSize*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, weightCuda, batchSize*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, outputCuda, batchSize*sizeof(float*)/sizeof(float));
    
    const float **inputB = (const float **)THCharTensor_data(inputHost);
    const float **weightB = (const float **)THCharTensor_data(weightHost);
    float **outputB = (float **)THCharTensor_data(outputHost);
    
    const float **inputB_d = (const float **)THCudaTensor_data(state, inputCuda);
    const float **weightB_d = (const float **)THCudaTensor_data(state, weightCuda);
    float **outputB_d = (float **)THCudaTensor_data(state, outputCuda);
    
    for (int i=0; i<batchSize; i++)
    {
      int inputIdx = THLongTensor_get1d(inputIndice, i) - 1;
      int outputIdx = THLongTensor_get1d(outputIndice, i) - 1;
      
      THCudaTensor_select(state, output_, output, 0, i);
      THCudaTensor_select(state, input_, input, 0, i);
      THCudaTensor_narrow(state, _weight_, weight, 1, inputIdx, inputWindowSize);
      THCudaTensor_narrow(state, weight_, _weight_, 0, outputIdx, outputWindowSize);
      
      inputB[i] = THCudaTensor_data(state, input_);
      weightB[i] = THCudaTensor_data(state, weight_);
      outputB[i] = THCudaTensor_data(state, output_);
    }
    
    if(hipMemcpy(inputB_d, inputB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(weightB_d, weightB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(outputB_d, outputB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    
                  
    stat = hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             outputWindowSize, 1, inputWindowSize,
                             &alpha, weightB_d, inputSize, 
                             inputB_d, inputWindowSize, 
                             &beta, outputB_d, outputWindowSize, 
                             batchSize);
    
    if (stat != HIPBLAS_STATUS_SUCCESS) 
      THError("hipblasSgemmBatched failed");
    
    
  }
  
  hipblasDestroy(handle);
  
  THCudaTensor_free(state, input_);
  THCudaTensor_free(state, weight_);
  THCudaTensor_free(state, _weight_);
  THCudaTensor_free(state, output_);
  THCudaTensor_free(state, bias_);

  return 1;
}



static int cunnx_WindowSparse_updateGradInput(lua_State *L)
{ 
  /* input, inputIndice, outputIndice, gradOutput*/
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THLongTensor *inputIndice = (THLongTensor*)luaT_checkudata(L, 3, "torch.LongTensor");
  THLongTensor *outputIndice = (THLongTensor*)luaT_checkudata(L, 4, "torch.LongTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  
  int batchedGemmMax = luaT_getfieldcheckint(L, 1, "batchedGemmMax");
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int outputWindowSize = luaT_getfieldcheckint(L, 1, "outputWindowSize");
  int batchSize, inputWindowSize;
  
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradInput", "torch.CudaTensor");
  THCudaTensor* gradOutput_, *weight_, *_weight_, *gradInput_;
  
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  
  float alpha = 1;
  float beta = 0;
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] <= inputSize, 2, "invalid input size"); 
  luaL_argcheck(L, inputIndice->nDimension == 1, 3, "1D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 1, 4, "1D(batch mode) tensor expected");
  
  THCudaTensor_resizeAs(state, gradInput, input); 
  
  batchSize = input->size[0];
  inputWindowSize = input->size[1];
    
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    THError("CUBLAS initialization failed");
    
  gradOutput_ = THCudaTensor_new(state);
  weight_ = THCudaTensor_new(state);
  _weight_ = THCudaTensor_new(state);
  gradInput_ = THCudaTensor_new(state);
  

  if (sqrt(inputWindowSize*outputWindowSize) > batchedGemmMax)
  {
    hipStream_t streams[WINDOWSPARSE_STREAMS];
    
    for (int i=0; i<WINDOWSPARSE_STREAMS; i++)
    {
      if (hipStreamCreate(&streams[i]) != hipSuccess)
        THError("error initializing stream");
    }
    hipDeviceSynchronize();
    
    for (int i=0; i<batchSize; i++)
    {
      hipblasSetStream(handle, streams[i%WINDOWSPARSE_STREAMS]);
      
      int inputIdx = THLongTensor_get1d(inputIndice, i) - 1;
      int outputIdx = THLongTensor_get1d(outputIndice, i) - 1;
      
      THCudaTensor_select(state, gradOutput_, gradOutput, 0, i);
      THCudaTensor_select(state, gradInput_, gradInput, 0, i);
      THCudaTensor_narrow(state, _weight_, weight, 1, inputIdx, inputWindowSize);
      THCudaTensor_narrow(state, weight_, _weight_, 0, outputIdx, outputWindowSize);
      
      stat = hipblasSgemv(handle, HIPBLAS_OP_N,  outputWindowSize, inputWindowSize,
                        &alpha, (const float*)THCudaTensor_data(state, weight_), inputSize,
                        (const float*)THCudaTensor_data(state, gradOutput_), 1,
                        &beta, THCudaTensor_data(state, gradInput_), 1);
                        
      if (stat != HIPBLAS_STATUS_SUCCESS) 
        THError("hipblasSgemv failed");
    }
    
    hipblasSetStream(handle, NULL);
    hipDeviceSynchronize();
  
    for (int i=0; i<WINDOWSPARSE_STREAMS; i++)
    {
      if (hipStreamDestroy(streams[i]) != hipSuccess)
        THError("error destroying stream");
    }
  
  }
  else
  {  
    THCharTensor *inputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "inputHost", "torch.CharTensor");
    THCharTensor *weightHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "weightHost", "torch.CharTensor");
    THCharTensor *outputHost = (THCharTensor*)luaT_getfieldcheckudata(L, 1, "outputHost", "torch.CharTensor");
    
    THCudaTensor *inputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputCuda", "torch.CudaTensor");
    THCudaTensor *weightCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weightCuda", "torch.CudaTensor");
    THCudaTensor *outputCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputCuda", "torch.CudaTensor");
    // put gradInput back on top of the stack
    gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_gradInput", "torch.CudaTensor");
    
    hipblasSetStream(handle, NULL);
    
    THCharTensor_resize1d(inputHost, batchSize*sizeof(float*));
    THCharTensor_resize1d(weightHost, batchSize*sizeof(float*));
    THCharTensor_resize1d(outputHost, batchSize*sizeof(float*));
    
    THCudaTensor_resize1d(state, inputCuda, batchSize*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, weightCuda, batchSize*sizeof(float*)/sizeof(float));
    THCudaTensor_resize1d(state, outputCuda, batchSize*sizeof(float*)/sizeof(float));
    
    float **gradInputB = (float **)THCharTensor_data(inputHost);
    const float **weightB = (const float **)THCharTensor_data(weightHost);
    const float **gradOutputB = (const float **)THCharTensor_data(outputHost);
    
    float **gradInputB_d = (float **)THCudaTensor_data(state, inputCuda);
    const float **weightB_d = (const float **)THCudaTensor_data(state, weightCuda);
    const float **gradOutputB_d = (const float **)THCudaTensor_data(state, outputCuda);
    
    for (int i=0; i<batchSize; i++)
    {
      int inputIdx = THLongTensor_get1d(inputIndice, i) - 1;
      int outputIdx = THLongTensor_get1d(outputIndice, i) - 1;
      
      THCudaTensor_select(state, gradOutput_, gradOutput, 0, i);
      THCudaTensor_select(state, gradInput_, gradInput, 0, i);
      THCudaTensor_narrow(state, _weight_, weight, 1, inputIdx, inputWindowSize);
      THCudaTensor_narrow(state, weight_, _weight_, 0, outputIdx, outputWindowSize);
      
      gradInputB[i] = THCudaTensor_data(state, gradInput_);
      weightB[i] = THCudaTensor_data(state, weight_);
      gradOutputB[i] = THCudaTensor_data(state, gradOutput_);
    }
    
    if(hipMemcpy(gradInputB_d, gradInputB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(weightB_d, weightB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
    if(hipMemcpy(gradOutputB_d, gradOutputB, sizeof(float*) * batchSize, hipMemcpyHostToDevice) != hipSuccess)
      THError("hipMemcpy failed");
                  
    stat = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             inputWindowSize, 1, outputWindowSize,
                             &alpha, weightB_d, inputSize, 
                             gradOutputB_d, outputWindowSize, 
                             &beta, gradInputB_d, inputWindowSize, 
                             batchSize);
    
    if (stat != HIPBLAS_STATUS_SUCCESS) 
      THError("hipblasSgemmBatched failed");
    
    
  }
  
  hipblasDestroy(handle);
  
  THCudaTensor_free(state, gradInput_);
  THCudaTensor_free(state, weight_);
  THCudaTensor_free(state, _weight_);
  THCudaTensor_free(state, gradOutput_);

  return 1;
}
  
__global__ void cunnx_WindowSparse_accGradParameters_kernel(
  float *gradWeight, float* gradBias, float *gradOutput, 
  float *input, float *inputIndice, float *outputIndice, 
  int inputWindowSize, int outputWindowSize, 
  int inputSize, int outputSize, float scale)
{
  __shared__ float buffer[WINDOWSPARSE_THREADS];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  
  int inputIdx = (int)inputIndice[k] - 1;
  int outputIdx = (int)outputIndice[k] - 1;
  
  float *input_k = input + k*inputWindowSize;
  float *gradOutput_k = gradOutput + k*outputWindowSize;
  float *gradWeight_k = gradWeight + outputIdx*inputSize + inputIdx;
  float *gradBias_k = gradBias + outputIdx;

  // addr weights (scalar-products)
  for (int i=tx; i<inputWindowSize; i+=i_step)
  {
    // copy input to buffer
    buffer[tx] = input_k[i]*scale;
  
    // multiply accumulate weights
    for (int j=0; j<outputWindowSize; j++)
      atomicAdd(&(gradWeight_k[j*inputSize + i]), gradOutput_k[j]*buffer[tx]);
  }
  
  // cadd bias i.e. multiply accumulate biases
  for (int j=tx; j<outputWindowSize; j+=i_step)
    atomicAdd(&(gradBias_k[j]), gradOutput_k[j]*scale);
}


static int cunnx_WindowSparse_accGradParameters(lua_State *L)
{ 
  /* input, inputIndice, outputIndice, gradOutput, scale */
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THLongTensor *inputIndice = (THLongTensor*)luaT_checkudata(L, 3, "torch.LongTensor");
  THLongTensor *outputIndice = (THLongTensor*)luaT_checkudata(L, 4, "torch.LongTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  float scale = luaL_optnumber(L, 6, 1);
  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int outputSize = luaT_getfieldcheckint(L, 1, "outputSize");
  int outputWindowSize = luaT_getfieldcheckint(L, 1, "outputWindowSize");
  int batchSize, inputWindowSize;
  
  // nOutputBlock x nInputBlock x outputSize x inputSize
  THCudaTensor *gradWeight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  
  THCudaTensor *inputIndiceCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "inputIndiceCuda", "torch.CudaTensor");
  THCudaTensor *outputIndiceCuda = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "outputIndiceCuda", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] <= inputSize, 2, "invalid input size"); 
  luaL_argcheck(L, inputIndice->nDimension == 1, 3, "1D(batch mode) tensor expected");
  luaL_argcheck(L, outputIndice->nDimension == 1, 4, "1D(batch mode) tensor expected");
  
  batchSize = input->size[0];
  inputWindowSize = input->size[1];
  
  THCudaTensor_resize1d(state, inputIndiceCuda, batchSize);
  THCudaTensor_resize1d(state, outputIndiceCuda, batchSize);
  
  THCudaTensor_copyLong(state, inputIndiceCuda, inputIndice);
  THCudaTensor_copyLong(state, outputIndiceCuda, outputIndice);
  
  /* call cudakernel */
  dim3 blocks(batchSize); // each cuda-block is an example
  dim3 threads(WINDOWSPARSE_THREADS);
  cunnx_WindowSparse_accGradParameters_kernel<<<blocks,threads>>>(
    THCudaTensor_data(state, gradWeight), THCudaTensor_data(state, gradBias), 
    THCudaTensor_data(state, gradOutput), THCudaTensor_data(state, input),
    THCudaTensor_data(state, inputIndiceCuda), THCudaTensor_data(state, outputIndiceCuda), 
    inputWindowSize, outputWindowSize, inputSize, outputSize, scale
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));  

  return 0;
}  
  
static const struct luaL_Reg cunnx_WindowSparse__ [] = {
  {"WindowSparse_updateOutput", cunnx_WindowSparse_updateOutput},
  {"WindowSparse_updateGradInput", cunnx_WindowSparse_updateGradInput},
  {"WindowSparse_accGradParameters", cunnx_WindowSparse_accGradParameters},
  {NULL, NULL}
};

static void cunnx_WindowSparse_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunnx_WindowSparse__, "nn");
  lua_pop(L,1);
}
