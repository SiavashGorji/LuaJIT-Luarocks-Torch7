#include "hip/hip_runtime.h"
#include "THCReduceApplyUtils.cuh"
#include "THCSortUtils.cuh"
#include "THCTensorCopy.h"
#include "THCTensorTypeUtils.cuh"

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

// Returns 2^(ceil(lg(n)) from Stanford bit twiddling hacks
unsigned long nextHighestPowerOf2(unsigned long n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  n |= n >> 32;
  n++;

  return n;
}

// `base` is the base address of a tensor
// For each slice (defined as a linear point of `out`, from 0 ->
// (sliceSize - 1) * sliceStride, we fill that slice from `0` to
// `sliceSize - 1`.
template <typename IndexType, int Dim>
__global__ void
fillSliceWithIndex(TensorInfo<float, IndexType> out,
                   IndexType totalSlices,
                   IndexType sliceSize,
                   IndexType sliceStride) {
  IndexType slice = getLinearBlockId<IndexType>();

  if (slice >= totalSlices) {
    return;
  }

  const unsigned long offset =
    IndexToOffset<float, IndexType, Dim>::get(slice, out);
  float* base = &out.data[offset];

  for (long i = threadIdx.x; i < sliceSize; i += blockDim.x) {
    // Torch indices are 1-based (hence the +1)
    base[i * sliceStride] = (float) i + 1.0f;
  }
}

void THCudaTensor_fillSliceWithIndex(THCState* state,
                                     THCudaTensor* t,
                                     int dim) {
  THCCheckTensorDims(state, t, 2);

  long inElements = THCudaTensor_nElement(state, t);
  long sliceSize = THCudaTensor_size(state, t, dim);
  long numSlices = inElements / sliceSize;

  dim3 grid;
  if (!THC_getGridFromTiles(numSlices, grid)) {
    THError("Slice to fill with indices is too large");
  }

  long maxThreads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  long numThreads = sliceSize;
  if (numThreads > maxThreads) {
    numThreads = maxThreads;
  }

  dim3 block(numThreads);

#define FILL_INDEX(T, DIM)                                       \
  fillSliceWithIndex<T, DIM>                                     \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(      \
      info, numSlices, sliceSize, info.strides[collapseDim])

  if (TensorUtils<THCudaTensor>::canUse32BitIndexMath(state, t)) {
    TensorInfo<float, unsigned int> info =
      getTensorInfo<THCudaTensor, unsigned int>(state, t);
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);

    if (info.isContiguous()) {
      FILL_INDEX(unsigned int, -2);
    } else {
      if (info.dims == 1) {
        FILL_INDEX(unsigned int, 1);
      } else if (info.dims == 2) {
        FILL_INDEX(unsigned int, 2);
      } else {
        FILL_INDEX(unsigned int, -1);
      }
    }
  } else {
    TensorInfo<float, unsigned long> info =
      getTensorInfo<THCudaTensor, unsigned long>(state, t);
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);

    // catch-all implementation
    FILL_INDEX(unsigned long, -1);
  }

#undef FILL_INDEX

  THCudaCheck(hipGetLastError());
}

// In alignment with default sort on a c++ map, this function
// will permute key and value tensors identically, and
// in such a way that the 'key' tensor is ordered numerically
THC_API void THCudaTensor_sortKeyValueInplace(THCState* state,
                                              THCudaTensor* key,
                                              THCudaTensor* value,
                                              int dim, bool dir) {
  THArgCheck(THCudaTensor_isSameSizeAs(state, key, value), 2,
             "Key tensor must have same size as value tensor");
  THCCheckTensorDims(state, key, 2);
  THCCheckTensorDims(state, value, 3);

  long inElements = THCudaTensor_nElement(state, key);
  long keySliceSize = THCudaTensor_size(state, key, dim);
  long keySlices = inElements / keySliceSize;

  if (THCudaTensor_nDimension(state, key) == 0) {
    // Zero-dim tensor; do nothing
    return;
  }

  // The amount of shared memory and block size is based on
  // 2^ceil(lg(n)); we choose that sorting implementation for a given
  // size.
  long ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);

  // FIXME: We'd have to find some other trick with Thrust to perform a
  // vectorized (key, value) sort by slice segment
  if (ceilPowerOf2 > 2048) {
    THError("sortKeyValueInplace only works for sizes <= 2048 at present");
  }

  int blockSize = (int) ceilPowerOf2 / 2;
  if (blockSize < 1) {
    blockSize = 1;
  }

  dim3 block(blockSize);

  // The grid is based on the number of independent slices that we
  // have to sort; one block per slice
  dim3 grid;
  if (!THC_getGridFromTiles(keySlices, grid)) {
    THError("Slice to sort is too large");
  }

#define HANDLE_CASE(TYPE, A, SIZE)                                      \
  if (dir) {                                                            \
    bitonicSortKVInPlace<float, float, A, -1, GTComp<float>, TYPE, SIZE> \
      <<<grid, block, 0, THCState_getCurrentStream(state)>>>(           \
        keyInfo,                                                        \
        keySlices,                                                      \
        (TYPE) keySliceSize,                                            \
        (TYPE) keyInfo.strides[collapseKeyDim],                         \
        valueInfo,                                                      \
        (TYPE) valueInfo.strides[collapseValueDim],                     \
        GTComp<float>());                                               \
  } else {                                                              \
    bitonicSortKVInPlace<float, float, A, -1, LTComp<float>, TYPE, SIZE> \
      <<<grid, block, 0, THCState_getCurrentStream(state)>>>(           \
        keyInfo,                                                        \
        keySlices,                                                      \
        (TYPE) keySliceSize,                                            \
        (TYPE) keyInfo.strides[collapseKeyDim],                         \
        valueInfo,                                                      \
        (TYPE) valueInfo.strides[collapseValueDim],                     \
        LTComp<float>());                                               \
  }

#define HANDLE_SORT_CASE(TYPE, A)                       \
  {                                                     \
    switch (ceilPowerOf2) {                             \
      case 2048:                                        \
      HANDLE_CASE(TYPE, A, 2048);                       \
      break;                                            \
      case 1024:                                        \
      HANDLE_CASE(TYPE, A, 1024);                       \
      break;                                            \
      case 512:                                         \
      HANDLE_CASE(TYPE, A, 512);                        \
      break;                                            \
      case 256:                                         \
      HANDLE_CASE(TYPE, A, 256);                        \
      break;                                            \
      case 128:                                         \
      HANDLE_CASE(TYPE, A, 128);                        \
      break;                                            \
      case 64:                                          \
      HANDLE_CASE(TYPE, A, 64);                         \
      break;                                            \
      case 32:                                          \
      HANDLE_CASE(TYPE, A, 32);                         \
      break;                                            \
      case 16:                                          \
      HANDLE_CASE(TYPE, A, 16);                         \
      break;                                            \
      case 8:                                           \
      HANDLE_CASE(TYPE, A, 8);                          \
      break;                                            \
      case 4:                                           \
      HANDLE_CASE(TYPE, A, 4);                          \
      break;                                            \
      case 2:                                           \
      HANDLE_CASE(TYPE, A, 2);                          \
      break;                                            \
      case 1:                                           \
      /* Nothing to do, data already sorted */          \
      break;                                            \
      default:                                          \
      assert(false);                                    \
    }                                                   \
  }

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  if (TensorUtils<THCudaTensor>::canUse32BitIndexMath(state, key)) {
    TensorInfo<float, unsigned int> keyInfo =
      getTensorInfo<THCudaTensor, unsigned int>(state, key);
    keyInfo.reduceDim(dim);
    int collapseKeyDim = keyInfo.collapseDims(dim);

    TensorInfo<float, unsigned int> valueInfo =
      getTensorInfo<THCudaTensor, unsigned int>(state, value);
    valueInfo.reduceDim(dim);
    int collapseValueDim = valueInfo.collapseDims(dim);

    if (keyInfo.isContiguous()) {
      HANDLE_SORT_CASE(unsigned int, -2);
    } else {
      switch (keyInfo.dims) {
        case 1:
          HANDLE_SORT_CASE(unsigned int, 1);
          break;
        case 2:
          HANDLE_SORT_CASE(unsigned int, 2);
          break;
        default:
          HANDLE_SORT_CASE(unsigned int, -1);
          break;
      }
    }
  } else {
    TensorInfo<float, unsigned long> keyInfo =
      getTensorInfo<THCudaTensor, unsigned long>(state, key);
    keyInfo.reduceDim(dim);
    int collapseKeyDim = keyInfo.collapseDims(dim);

    TensorInfo<float, unsigned long> valueInfo =
      getTensorInfo<THCudaTensor, unsigned long>(state, value);
    valueInfo.reduceDim(dim);
    int collapseValueDim = valueInfo.collapseDims(dim);

    // long case is rare, just instantiate these versions
    if (keyInfo.isContiguous()) {
      HANDLE_SORT_CASE(unsigned long, -2);
    } else {
      HANDLE_SORT_CASE(unsigned long, -1);
    }
  }
#undef HANDLE_CASE
#undef HANDLE_SORT_CASE
#undef HANDLE_A_CASE

  THCudaCheck(hipGetLastError());
}

// For slice sorting in Thrust; extracts a slice index from a linear
// index and uses that for comparison
struct SliceComp {
  SliceComp(int size) : sliceSize(size) {}

  __device__ bool operator()(const int& a, const int& b) const {
    // Since the slices are guaranteed to be innermost, the segment is
    // just via integer division
    int segA = a / sliceSize;
    int segB = b / sliceSize;
    return segA < segB;
  }

  const int sliceSize;
};

// For sorting in Thurst; extracts a within-slice index from a linear index
struct GlobalIndexToPerSliceIndex {
  GlobalIndexToPerSliceIndex(int size) : sliceSize(size) {}

  __device__ inline void operator()(int& v) const {
    // Thrust is operating on this index array as an array of type
    // int, but to Torch it should be a float array.
    v = __float_as_int((float) (v % sliceSize + 1));
  }

  const int sliceSize;
};

void sortViaThrust(THCState* state,
                   THCudaTensor* sorted,
                   THCudaTensor* indices,
                   THCudaTensor* input,
                   int dim, bool dir) {
  long nDims = THCudaTensor_nDimension(state, input);

  long totalElements = THCudaTensor_nElement(state, input);
  long sliceSize = THCudaTensor_size(state, input, dim);
  long sliceStride = THCudaTensor_stride(state, input, dim);

  // We perform a vectorized segmented sort in Thrust.
  // Say we are sorting a (2, 3) tensor. We have in flattened form:
  // values 0.4 1.2 5.3 6.2 1.3 2.3
  // indices  0   1   2   3   4   5
  // where indices is a global index (across all slices)

  // First we sort by values, globally:
  // values 6.2 5.3 2.3 1.2 1.3 0.4
  // indices  3   2   5   1   4   0

  // Then we stable sort by segment, which is index / 3:
  // values 5.3 1.2 0.4 6.2 2.3 1.3
  // indices  2   1   0   3   5   4

  // Then we translate the global index to a per-slice Lua index
  // (index % 3) + 1:
  // values 5.3 1.2 0.4 6.2 2.3 1.3
  // indices  3   2   1   1   3   2

  // This method can only work if the slice we are sorting (`dim`) is
  // innermost, and both values and indices are contiguous. We do this
  // by re-arranging the input into this form as needed, which will
  // unfortunately allocate memory if the request is not in this form.
  // Vectorized sort is slower than iterated sort if the number of
  // slices is small (since we're sorting twice, instead of invoking a
  // smaller sort `numSlices` times), but the Thrust sort
  // implementation here is a catch-all, so we're not looking for
  // efficiency, but instead correctness.
  THCudaTensor_copy(state, sorted, input);
  THCudaTensor* trKeys = THCudaTensor_newWithTensor(state, sorted);
  THCudaTensor* trIndices = THCudaTensor_newWithTensor(state, indices);

  // Transpose dim to innermost
  if (dim != nDims - 1) {
    THCudaTensor_transpose(state, trKeys, NULL, dim, nDims - 1);
    THCudaTensor_transpose(state, trIndices, NULL, dim, nDims - 1);
  }

  // Thrust must operate on a contiguous layout
  THCudaTensor* trContigKey = THCudaTensor_newContiguous(state, trKeys);
  THCudaTensor* trContigIndices = THCudaTensor_newContiguous(state, trIndices);

  THCudaTensor_free(state, trKeys);
  THCudaTensor_free(state, trIndices);

  thrust::device_ptr<float> keyIter(THCudaTensor_data(state, trContigKey));

  // Since we are composing a global index across all segments rather
  // than a per-segment index, we treat the memory as int so we don't
  // have problems sorting slices < 2^24 but where the entire tensor
  // has more than 2^24 elements
  thrust::device_ptr<int>
    indexIter((int*) THCudaTensor_data(state, trContigIndices));

  // Fill the indices with a global index across all slices
  thrust::counting_iterator<int> countIter(0);

  thrust::copy(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    countIter, countIter + totalElements, indexIter);

  // First, we sort globally (across all slices) according to key
  // (the values we're sorting)
  if (dir) {
    thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      keyIter, keyIter + totalElements, indexIter, thrust::greater<float>());
  } else {
    thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      keyIter, keyIter + totalElements, indexIter, thrust::less<float>());
  }

  // Then, re-sort according to slice that each index is
  // in. This completes the segment sort in Thrust, since we're
  // stably sorting here, preserving the relative order of values
  // per each slice
  thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    indexIter, indexIter + totalElements, keyIter,
    SliceComp(sliceSize));

  // Translate the global integer 0-based index to a per-slice float
  // Lua index
  thrust::for_each(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    indexIter, indexIter + totalElements,
    GlobalIndexToPerSliceIndex(sliceSize));

  // Reverse the transposition as needed
  if (dim != nDims - 1) {
    THCudaTensor_transpose(state, trContigKey, NULL, dim, nDims - 1);
    THCudaTensor_transpose(state, trContigIndices, NULL, dim, nDims - 1);
  }

  // Then copy back to the expected output
  THCudaTensor_freeCopyTo(state, trContigKey, sorted);
  THCudaTensor_freeCopyTo(state, trContigIndices, indices);
}

THC_API void THCudaTensor_sort(THCState* state,
                               THCudaTensor *sorted,
                               THCudaTensor *indices,
                               THCudaTensor *input,
                               int dim, int order) {
  THAssert(THCudaTensor_checkGPU(state, 3, sorted, indices, input));
  THCCheckTensorDims(state, sorted, 2);
  THCCheckTensorDims(state, indices, 3);
  THCCheckTensorDims(state, input, 4);

  // Make sure sufficient output space is allocated
  THCudaTensor_resizeAs(state, sorted, input);
  THCudaTensor_resizeAs(state, indices, input);

  // How large are the slices that we are sorting?
  long sliceSize = THCudaTensor_size(state, input, dim);

  // We're using THCudaTensor to write out indices, so if the slice
  // size that we're sorting has more elements than can be
  // represented in fp32, warn the user
  // FIXME: this isn't a real restriction of either our code or of
  // Thrust, but we have to switch to a CUDA long tensor to support
  // larger slice sizes. Otherwise the indices will contain garbage.
  THArgCheck(sliceSize <= (long) FLOAT32_MAX_CONSECUTIVE_INT, 5,
             "The sort dimension exceeds single-precision float "
             "consecutive integer precision size (2^24), since float "
             "is used for indices");

  if (sliceSize <= 2048) {
    // Fill `indices` (the values) with the
    // slice-relative index.
    THCudaTensor_fillSliceWithIndex(state, indices, dim);

    // We sort k/v pairs in-place; copy unsorted input to output
    THCudaTensor_copy(state, sorted, input);

    // Sort using our in-place k/v kernel that supports arbitrary
    // layout
    THCudaTensor_sortKeyValueInplace(state, sorted, indices, dim, order);
  } else {
    // Otherwise, fall back upon Thrust, which handles all other cases
    // (potentially slowly, with extra copies/memory allocations)
    sortViaThrust(state, sorted, indices, input, dim, (bool) order);
  }

  THCudaCheck(hipGetLastError());
}
